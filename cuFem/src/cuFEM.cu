#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <iostream>

#include "cuFEM.h"

void QRwithCusolver(int nnz, int n, double* h_csrValA, double* h_b, double* h_x, int* h_csrRowPtrA, int* h_csrColIndA)
{
    // QR direct way with cusolver
    hipsolverSpHandle_t handle;
    hipsolverStatus_t solverStatus;

    hipsparseStatus_t spStatus;
    hipsparseMatDescr_t descrA;

    solverStatus = hipsolverSpCreate(&handle);
    spStatus = hipsparseCreateMatDescr(&descrA);

    double* d_csrValA, *d_b, *d_x;
    int* d_csrRowPtrA, *d_csrColIndA;
    hipMalloc((void**)&d_csrValA, nnz * sizeof(double));
    hipMalloc((void**)&d_b, n * sizeof(double));
    hipMalloc((void**)&d_x, n * sizeof(double));
    hipMalloc((void**)&d_csrRowPtrA, (n+1) * sizeof(int));
    hipMalloc((void**)&d_csrColIndA, n * sizeof(int));

    hipMemcpy(d_csrValA, h_csrValA, nnz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, (n+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, n * sizeof(int), hipMemcpyHostToDevice);

    std::cout << "----------solving with cusolver---------------" << std::endl;
    double tol = 1e-16;
    int reorder = 1;
    int sigularity = 0;
    solverStatus = hipsolverSpDcsrlsvqr(handle, n, nnz, descrA, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_b, tol, reorder, d_x, &sigularity);

    std::cout << "----------END solving with cusolver---------------" << std::endl;

    hipMemcpy(h_x, d_x, n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_csrRowPtrA);
    hipFree(d_csrValA);
    hipFree(d_csrColIndA);
    hipFree(d_b);
    hipFree(d_x);
    hipsolverSpDestroy(handle);
    
}
